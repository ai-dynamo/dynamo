// SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

// Error checking macro
#define CUDA_CHECK(call)                                                                            \
  do {                                                                                              \
    hipError_t error = call;                                                                       \
    if (error != hipSuccess) {                                                                     \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
      return error;                                                                                 \
    }                                                                                               \
  } while (0)

// Number of elements to process per thread
#define ELEMENTS_PER_THREAD 4

// Optimized kernel that processes multiple elements per thread
__global__ void
copy_blocks_kernel(
    const void* src_data, void* dst_data, const int* src_block_ids, const int* dst_block_ids, int num_blocks,
    int kv_size, int block_size, int heads_per_rank, int head_size, int elem_size, size_t src_tp_stride,
    size_t src_block_stride, size_t src_pos_stride, size_t src_head_stride, size_t dst_tp_stride,
    size_t dst_block_stride, size_t dst_pos_stride, size_t dst_head_stride)
{
  // Get global thread index
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Calculate total elements per block
  int elements_per_block = block_size * heads_per_rank * head_size;

  // Total elements to process
  int total_elements = elements_per_block * num_blocks * kv_size;

  // Calculate the starting element index for this thread
  int start_element = thread_idx * ELEMENTS_PER_THREAD;

  // Process multiple elements per thread
  for (int e = 0; e < ELEMENTS_PER_THREAD; e++) {
    // Current element index
    int elem_idx = start_element + e;

    // Check if this element is within bounds
    if (elem_idx >= total_elements) {
      return;  // No more elements to process
    }

    // Calculate which element we're processing
    int combined_idx = elem_idx / elements_per_block;
    int element_offset = elem_idx % elements_per_block;

    // Split combined_idx into rank and block
    int rank = combined_idx / num_blocks;       // KV rank (0 or 1)
    int block_idx = combined_idx % num_blocks;  // Block index in the mapping list

    // Bounds check on block indices
    if (block_idx < 0 || block_idx >= num_blocks) {
      continue;  // Skip this element
    }

    // Get source and destination block IDs
    int src_block_id = src_block_ids[block_idx];
    int dst_block_id = dst_block_ids[block_idx];

    // Calculate position indices
    int pos = element_offset / (heads_per_rank * head_size);
    int remaining = element_offset % (heads_per_rank * head_size);
    int head = remaining / head_size;
    int head_offset = remaining % head_size;

    // Bounds check on all indices
    if (pos >= block_size || head >= heads_per_rank || head_offset >= head_size) {
      continue;  // Skip this element
    }

    // Calculate source offset using provided strides
    size_t src_offset = rank * src_tp_stride + src_block_id * src_block_stride + pos * src_pos_stride +
                        head * src_head_stride + head_offset;

    // Calculate destination offset
    size_t dst_offset = rank * dst_tp_stride + dst_block_id * dst_block_stride + pos * dst_pos_stride +
                        head * dst_head_stride + head_offset;

    // Perform type-optimized copy based on element size
    if (elem_size == 2) {
      // For 16-bit elements (half/bfloat16/uint16)
      const uint16_t* src_ptr = (const uint16_t*)src_data + src_offset;
      uint16_t* dst_ptr = (uint16_t*)dst_data + dst_offset;
      *dst_ptr = *src_ptr;
    } else if (elem_size == 4) {
      // For 32-bit elements (float/int32)
      const uint32_t* src_ptr = (const uint32_t*)src_data + src_offset;
      uint32_t* dst_ptr = (uint32_t*)dst_data + dst_offset;
      *dst_ptr = *src_ptr;
    } else if (elem_size == 8) {
      // For 64-bit elements (double/int64)
      const uint64_t* src_ptr = (const uint64_t*)src_data + src_offset;
      uint64_t* dst_ptr = (uint64_t*)dst_data + dst_offset;
      *dst_ptr = *src_ptr;
    } else {
      // For other element sizes, copy byte by byte
      const char* src_bytes = (const char*)src_data + src_offset * elem_size;
      char* dst_bytes = (char*)dst_data + dst_offset * elem_size;

      // Copy element using proper size
      for (int i = 0; i < elem_size; i++) {
        dst_bytes[i] = src_bytes[i];
      }
    }
  }
}

// Host-callable function
extern "C" hipError_t
copy_blocks(
    const void* src_data, void* dst_data, const int* src_block_ids, int num_src_blocks, const int* dst_block_ids,
    int num_dst_blocks, int src_n_blocks, int dst_n_blocks, int kv_size, int block_size, int heads_per_rank,
    int head_size, int elem_size, size_t src_tp_stride, size_t src_block_stride, size_t src_pos_stride,
    size_t src_head_stride, size_t dst_tp_stride, size_t dst_block_stride, size_t dst_pos_stride,
    size_t dst_head_stride)
{
  // Validate inputs
  if (src_data == NULL || dst_data == NULL) {
    fprintf(stderr, "NULL data pointers\n");
    return hipErrorInvalidValue;
  }

  if (src_block_ids == NULL || dst_block_ids == NULL) {
    fprintf(stderr, "NULL block ID pointers\n");
    return hipErrorInvalidValue;
  }

  if (num_src_blocks != num_dst_blocks || num_src_blocks <= 0) {
    fprintf(stderr, "Block list issue: src=%d, dst=%d\n", num_src_blocks, num_dst_blocks);
    return hipErrorInvalidValue;
  }

  if (kv_size <= 0 || block_size <= 0 || heads_per_rank <= 0 || head_size <= 0 || elem_size <= 0) {
    fprintf(
        stderr, "Invalid dimensions: tp=%d, block=%d, heads=%d, head_size=%d, elem=%d\n", kv_size, block_size,
        heads_per_rank, head_size, elem_size);
    return hipErrorInvalidValue;
  }

  // Copy block IDs to device
  int* d_src_blocks = NULL;
  int* d_dst_blocks = NULL;

  // TODO: Create class/struct with preallocated memory for block_ids and two cuda events
  CUDA_CHECK(hipMalloc(&d_src_blocks, num_src_blocks * sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_dst_blocks, num_dst_blocks * sizeof(int)));

  CUDA_CHECK(hipMemcpy(d_src_blocks, src_block_ids, num_src_blocks * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_dst_blocks, dst_block_ids, num_dst_blocks * sizeof(int), hipMemcpyHostToDevice));

  // Calculate grid dimensions with ELEMENTS_PER_THREAD adjustment
  int elements_per_block = block_size * heads_per_rank * head_size;
  int total_elements = elements_per_block * num_src_blocks * kv_size;

  // Adjust grid size to account for multiple elements per thread
  int total_threads = (total_elements + ELEMENTS_PER_THREAD - 1) / ELEMENTS_PER_THREAD;
  int cuda_block_size = 256;
  int grid_size = (total_threads + cuda_block_size - 1) / cuda_block_size;

  // Validate grid size
  if (grid_size <= 0) {
    fprintf(stderr, "Invalid grid size: %d\n", grid_size);
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
    return hipErrorInvalidValue;
  }

  // Print debug information
  printf("Starting kernel: blocks=%d, threads=%d, total elements=%d\n", grid_size, cuda_block_size, total_elements);
  printf("Elements per thread: %d, Total threads: %d\n", ELEMENTS_PER_THREAD, total_threads);
  printf(
      "Dimensions: tp=%d, blocks=%d, size=%d, heads=%d, headsize=%d, elemsize=%d\n", kv_size, num_src_blocks,
      block_size, heads_per_rank, head_size, elem_size);

  // Launch kernel
  copy_blocks_kernel<<<grid_size, cuda_block_size>>>(
      src_data, dst_data, d_src_blocks, d_dst_blocks, num_src_blocks, kv_size, block_size, heads_per_rank, head_size,
      elem_size, src_tp_stride, src_block_stride, src_pos_stride, src_head_stride, dst_tp_stride, dst_block_stride,
      dst_pos_stride, dst_head_stride);

  // Check for kernel errors immediately
  hipError_t kernel_error = hipGetLastError();
  if (kernel_error != hipSuccess) {
    fprintf(stderr, "Kernel execution error: %s\n", hipGetErrorString(kernel_error));
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
    return kernel_error;
  }

  // Wait for completion
  CUDA_CHECK(hipDeviceSynchronize());

  // Clean up
  hipFree(d_src_blocks);
  hipFree(d_dst_blocks);

  printf("Kernel execution completed successfully\n");
  return hipSuccess;
}


// TODO: Refactor the driver code to take pointers for the device block_id arrays
// TODO: Maintain a blocking driver, but then also provide a non-blocking driver
//
// We will have N copies of the BlockCopyControl struct which we will put in a reusable
// pool. Acquiring a BlockCopyControl will let you perform a copy for a kv attention layer.
//
// From rust or python we'll execute this on a thread allowed to block. We'll await the
// cuda event for completion and report the return code on the driver.
//
// TODO: decide whether or not we need a pool of streams or use a single stream.
//
// We should be able to decouple this from the forward pass. The only condition is that
// a new forward pass can not start until the last copy has completed.
//
// To that end, we might want to tie this copy kernel to the stream used for the forward pass.
struct BlockCopyControl {
  int* d_src_blocks;
  int* d_dst_blocks;
  hipEvent_t start_event;
  hipEvent_t stop_event;

  BlockCopyControl(int num_blocks);
  ~BlockCopyControl();
};

BlockCopyControl::BlockCopyControl(int num_blocks)
{
  hipError_t status;
  status = hipMalloc(&d_src_blocks, num_blocks * sizeof(int));
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    return;
  }

  status = hipMalloc(&d_dst_blocks, num_blocks * sizeof(int));
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    hipFree(d_src_blocks);
    return;
  }

  status = hipEventCreate(&start_event);
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
  }

  status = hipEventCreate(&stop_event);
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
  }
}

BlockCopyControl::~BlockCopyControl()
{
  hipFree(d_src_blocks);
  hipFree(d_dst_blocks);
}
